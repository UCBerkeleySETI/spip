#include "hip/hip_runtime.h"
/***************************************************************************
 *
 *   Copyright (C) 2016 Andrew Jameson
 *   Licensed under the Academic Free License version 2.1
 *
 ***************************************************************************/

#include "spip/AdaptiveFilterCUDA.h"
#include "spip/Error.h"

#include <iostream>
#include <cmath>
#include <hip/hip_complex.h>
#include <cstdio>

using namespace std;

// compute a sum of a hipFloatComplex across a warp
__inline__ __device__
hipFloatComplex warpReduceSumFC(hipFloatComplex val) 
{
  for (int offset = warpSize/2; offset > 0; offset /= 2)
  {
    val.x += __shfl_down(val.x, offset);
    val.y += __shfl_down(val.y, offset);
  }
  return val;
}

// compute a sum of a float across a block
__inline__ __device__
hipFloatComplex blockReduceSumFC(hipFloatComplex val) 
{
  __shared__ hipFloatComplex shared[32]; // shared mem for 32 partial sums

  int lane = threadIdx.x % warpSize;
  int wid = threadIdx.x / warpSize;

  val = warpReduceSumFC(val);     // Each warp performs partial reduction

  if (lane==0) shared[wid] = val; // Write reduced value to shared memory

  __syncthreads();              // Wait for all partial reductions

  //read from shared memory only if that warp existed
  val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : make_hipFloatComplex(0, 0);

  if (wid==0) val = warpReduceSumFC(val); //Final reduce within first warp

  return val;
}


// SFPT implementation of adaptive filter algorithm
__global__ void AdaptiveFilterKernel_SFPT (hipFloatComplex * in, hipFloatComplex * out, hipFloatComplex * gains,
                                           uint64_t nloops, 
                                           uint64_t in_sig_stride, uint64_t in_chan_stride, 
                                           uint64_t out_sig_stride, uint64_t out_chan_stride,
                                           uint64_t pol_stride, unsigned ref_pol)
{
  // there is one complex gain per block
  __shared__ hipFloatComplex g;

  const unsigned ichanpol = (blockIdx.z * gridDim.y) + blockIdx.y;

  // get the first thread to read the previously computed complex gain
  if (threadIdx.x == 0)
  {
    g = gains[ichanpol];
  }

  // ensure all threads have the complex gain
  __syncthreads();

  // offsets for input, reference and output
  uint64_t idx = (blockIdx.x * in_sig_stride)  + (blockIdx.y * in_chan_stride)  + (blockIdx.z * pol_stride) + threadIdx.x;
  uint64_t rdx = (blockIdx.x * in_sig_stride)  + (blockIdx.y * in_chan_stride)  + (ref_pol * pol_stride)    + threadIdx.x;
  uint64_t odx = (blockIdx.x * out_sig_stride) + (blockIdx.y * out_chan_stride) + (blockIdx.z * pol_stride) + threadIdx.x;

  //if (threadIdx.x == 0 || threadIdx.x == 1023)
  //  printf ("[%d][%d] ichanpol=%u idx=%lu\n", blockIdx.x, threadIdx.x, ichanpol, idx);

  __shared__ float normalized_factor;
  float previos_factor;

  for (unsigned iloop=0; iloop<nloops; iloop++)
  {
    // read the reference antenna value
    hipFloatComplex r = in[rdx];
    // read the astronomy antenna value
    hipFloatComplex a = in[idx];

    ///////////////////////////////////
    hipFloatComplex pa = hipCmulf(a, hipConjf(a));
    hipFloatComplex pr = hipCmulf(r, hipConjf(r));

    pa = blockReduceSumFC(pa);
    pr = blockReduceSumFC(pr);

    if (threadIdx.x == 0)
    {
      // normalise
      float pn = pa.x / blockDim.x + pr.x / blockDim.x;
      //normalized_power.y = pa.y / blockDim.y + pr.y / blockDim.y;

    float current_factor = pn;


    if(iloop == 0)
    {
      normalized_factor = 0.999 * current_factor + 0.001 *current_factor;
    }
    else
    {
      normalized_factor = 0.999 * previos_factor + 0.001 *current_factor;
    }

    previos_factor = current_factor;

    }

    // ensure pn are common across the block
    __syncthreads();

//////////////////////////////////

    // compute complex conjugate f = [gain * ref]
    hipFloatComplex f = hipCmulf(g, r);

    // subtract from the astronomy signal [af = ast - f]
    hipFloatComplex af = hipCsubf(a, f);

    // compute correlation [corr = af * conj(ref)]
    hipFloatComplex corr = hipCmulf(af, hipConjf(r));

    corr.x /= normalized_factor;
    corr.y /= normalized_factor;

    // sum corr across the block
    corr = blockReduceSumFC(corr);

    // get thread 0 to compute the new gains
    if (threadIdx.x == 0)
    {
      // normalise
      corr.x /= blockDim.x;
      corr.y /= blockDim.x;

      // compute new gain
      const float epsilon = 0.1;
      g.x = (corr.x * epsilon) + g.x;
      g.y = (corr.y * epsilon) + g.y;
    }

    // ensure gains are common across the block
    __syncthreads();

    // now that the gain is updated, for this current block
    f = hipCmulf (g, r);

    // and subtract from the astronomy signal
    af = hipCsubf(a, f);

    // write the output global memory
    out[odx] = af;
  
    // increment to the next filter
    idx += blockDim.x;
    rdx += blockDim.x;
    odx += blockDim.x;
  }

  // update the gains
  if (threadIdx.x == 0)
  {
    gains[ichanpol] = g;
  }
}


spip::AdaptiveFilterCUDA::AdaptiveFilterCUDA (hipStream_t _stream, string dir) : AdaptiveFilter (dir)
{
  stream = _stream;
}

spip::AdaptiveFilterCUDA::~AdaptiveFilterCUDA ()
{
}

// configure the pipeline prior to runtime
void spip::AdaptiveFilterCUDA::configure (spip::Ordering output_order)
{
  // TODO implement a ContainerCUDADeviceFileWrite class [Nuer]
  if (!gains)
    gains = new spip::ContainerCUDADevice ();

  spip::AdaptiveFilter::configure (output_order);
}

// convert to antenna minor order
void spip::AdaptiveFilterCUDA::transform_TSPF()
{
  if (verbose)
    std::cerr << "spip::AdaptiveFilterCUDA::transform_TSPF ()" << endl;
  throw Error (InvalidState, "spip::AdaptiveFilterCUDA::transform_TSPF", "not implemented");
}

void spip::AdaptiveFilterCUDA::transform_SFPT()
{
  if (verbose)
    cerr << "spip::AdaptiveFilterCUDA::transform_SFPT ()" << endl;

  // pointers to the buffers for in, rfi and out
  hipFloatComplex * in = (hipFloatComplex *) input->get_buffer();
  hipFloatComplex * out = (hipFloatComplex *) output->get_buffer();
  hipFloatComplex * gai = (hipFloatComplex *) gains->get_buffer();

  dim3 blocks (nsignal, nchan, npol);
  unsigned nthread = 1024;
  if (nthread > filter_update_time)
    nthread = filter_update_time;

  unsigned nloops = ndat / nthread;
  if (ndat % nthread != 0)
    nloops++;

  if (verbose)
  {
    cerr << "spip::AdaptiveFilterCUDA::transform_SFPT nsignal=" << nsignal << " nchan=" << nchan << " npol=" << npol
<< endl;
    cerr << "spip::AdaptiveFilterCUDA::transform_SFPT ndat=" << ndat << " nthread=" << nthread << " nloops=" << nloops << " kernels" << endl;
  }

  uint64_t pol_stride = ndat;
  uint64_t in_chan_stride = npol * pol_stride;
  uint64_t in_sig_stride  = nchan * in_chan_stride;
  uint64_t out_chan_stride = out_npol * pol_stride;
  uint64_t out_sig_stride  = nchan * out_chan_stride;

  AdaptiveFilterKernel_SFPT<<<blocks, nthread, 0, stream>>>(in, out, gai, nloops, in_sig_stride, in_chan_stride, out_sig_stride, out_chan_stride, pol_stride, ref_pol);

  if (verbose)
    cerr << "spip::AdaptiveFilterCUDA::transform_SFPT kernels complete" << endl;
}

void spip::AdaptiveFilterCUDA::write_gains ()
{
  throw Error (InvalidState, "spip::AdaptiveFilterCUDA::write_gains", "not implemented");
}
