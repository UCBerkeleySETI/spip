#include "hip/hip_runtime.h"
/***************************************************************************
 *
 *   Copyright (C) 2016 Andrew Jameson
 *   Licensed under the Academic Free License version 2.1
 *
 ***************************************************************************/

#include "spip/AdaptiveFilterCUDA.h"
#include "spip/Error.h"
#include "spip/CUDAError.h"

#include <iostream>
#include <cmath>
#include <hip/hip_complex.h>
#include <cstdio>

using namespace std;
__inline__ __device__ 
float squareLawDetect(hipFloatComplex val)
{
  return val.x * val.x + val.y * val.y;
}

// compute a sum of a hipFloatComplex across a warp
__inline__ __device__
hipFloatComplex warpReduceSumFC(hipFloatComplex val) 
{
  for (int offset = warpSize/2; offset > 0; offset /= 2)
  {
    val.x += __shfl_down(val.x, offset);
    val.y += __shfl_down(val.y, offset);
  }
  return val;
}

// compute a sum of a hipFloatComplex across a block
__inline__ __device__
hipFloatComplex blockReduceSumFC(hipFloatComplex val) 
{
  __shared__ hipFloatComplex shared[32]; // shared mem for 32 partial sums

  int lane = threadIdx.x % warpSize;
  int wid = threadIdx.x / warpSize;

  val = warpReduceSumFC(val);     // Each warp performs partial reduction

  if (lane==0) shared[wid] = val; // Write reduced value to shared memory

  __syncthreads();              // Wait for all partial reductions

  //read from shared memory only if that warp existed
  val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : make_hipFloatComplex(0, 0);

  if (wid==0) val = warpReduceSumFC(val); //Final reduce within first warp

  return val;
}

// compute a sum of a float across a warp
__inline__ __device__
float warpReduceSumF(float val)
{
  for (int offset = warpSize/2; offset > 0; offset /= 2)
  {
    val += __shfl_down(val, offset);
  }
  return val;
}

// compute a sum of a float across aa block
__inline__ __device__
float blockReduceSumF(float val)
{
  __shared__ float shared[32]; // shared mem for 32 partial sums

  int lane = threadIdx.x % warpSize;
  int wid = threadIdx.x / warpSize;

  val = warpReduceSumF(val);     // Each warp performs partial reduction

  if (lane==0) shared[wid] = val; // Write reduced value to shared memory

  __syncthreads();              // Wait for all partial reductions

  //read from shared memory only if that warp existed
  val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0;

  if (wid==0) val = warpReduceSumF(val); //Final reduce within first warp

  return val;
}



// SFPT implementation of adaptive filter algorithm
__global__ void AdaptiveFilterKernel_SFPT (hipFloatComplex * in, hipFloatComplex * out, hipFloatComplex * gains, 
                                           float * dirty, float * cleaned, float * norms,
                                           uint64_t nloops, bool processed_first_block,
                                           uint64_t in_sig_stride, uint64_t in_chan_stride, 
                                           uint64_t out_sig_stride, uint64_t out_chan_stride,
                                           uint64_t pol_stride, unsigned ref_pol)
{
  // there is one complex gain per block
  __shared__ hipFloatComplex g;
  
  // Block X = Signal
  // Block Y = Channel
  // Block Z = Polarisation
  // Gains are nominally stored in TSPF where T == 1, so SPF
  const unsigned isigchanpol = (blockIdx.x * gridDim.y * gridDim.z) + 
                               (blockIdx.z * gridDim.y) + 
                               blockIdx.y;

  // get the first thread to read the previously computed complex gain
  if (threadIdx.x == 0)
  {
    g = gains[isigchanpol];
  }

  // ensure all threads have the complex gain
  __syncthreads();

  // the reference polarisation could be (unfortunately) not the final ones
  const unsigned ast_pol = (blockIdx.z < ref_pol) ? blockIdx.z : blockIdx.z + 1;

  // offsets for input, reference and output, stored in SFPT
  uint64_t idx = (blockIdx.x * in_sig_stride)  + (blockIdx.y * in_chan_stride)  + (ast_pol * pol_stride)    + threadIdx.x;
  uint64_t rdx = (blockIdx.x * in_sig_stride)  + (blockIdx.y * in_chan_stride)  + (ref_pol * pol_stride)    + threadIdx.x;
  uint64_t odx = (blockIdx.x * out_sig_stride) + (blockIdx.y * out_chan_stride) + (blockIdx.z * pol_stride) + threadIdx.x;

  __shared__ float normalized_factor;

  for (unsigned iloop=0; iloop<nloops; iloop++)
  {
    // read the reference antenna value
    hipFloatComplex r = in[rdx];

    // read the astronomy antenna value
    hipFloatComplex a = in[idx];

    // power in in the astronomy and reference signals
    float pa = blockReduceSumF(squareLawDetect(a));
    float pr = blockReduceSumF(squareLawDetect(r));

    if (threadIdx.x == 0)
    {
      // normalise by the number of values used
      float current_factor = (pa / blockDim.x) + (pr / blockDim.x);

      if (processed_first_block || iloop > 0)
        normalized_factor = (0.999 * norms[isigchanpol]) + (0.001 * current_factor);
      else
        normalized_factor = current_factor;
      norms[isigchanpol] = normalized_factor;
    }

    // ensure normalized factor is shared across the block
    __syncthreads();

    // compute complex conjugate f = [gain * ref]
    hipFloatComplex f = hipCmulf(g, r);

    // subtract from the astronomy signal [af = ast - f]
    hipFloatComplex af = hipCsubf(a, f);

    // compute correlation [corr = af * conj(ref)]
    hipFloatComplex corr = hipCmulf(af, hipConjf(r));

    corr.x /= normalized_factor;
    corr.y /= normalized_factor;

    // sum corr across the block
    corr = blockReduceSumFC(corr);

    // get thread 0 to compute the new gains
    if (threadIdx.x == 0)
    {
      // normalise
      corr.x /= blockDim.x;
      corr.y /= blockDim.x;

      // compute new gain
      const float epsilon = 0.001;
      g.x = (corr.x * epsilon) + g.x;
      g.y = (corr.y * epsilon) + g.y;
    }

    // ensure gains are common across the block
    __syncthreads();

    // now that the gain is updated, for this current block
    f = hipCmulf (g, r);

    // and subtract from the astronomy signal
    af = hipCsubf(a, f);

    // write the output global memory
    out[odx] = af;

    //  save dirty and clean signals in first loop only
    if (iloop == 0)
    {
      float cleaned_power_sum = squareLawDetect(af);
      cleaned_power_sum = blockReduceSumF(cleaned_power_sum);
      if (threadIdx.x == 0)
      {
        cleaned[isigchanpol] = cleaned_power_sum / blockDim.x;
        dirty[isigchanpol] = pa / blockDim.x;
      }
    }
  
    // increment to the next filter
    idx += blockDim.x;
    rdx += blockDim.x;
    odx += blockDim.x;
  }

  // update the gains
  if (threadIdx.x == 0)
  {
    gains[isigchanpol] = g;
  }
}

// FPT implementation of adaptive filter algorithm
__global__ void AdaptiveFilterKernel_FPT (const __restrict__ hipFloatComplex * in, hipFloatComplex * out, hipFloatComplex * gains, 
                                          float * dirty, float * cleaned, float * norms,
                                          uint64_t nloops, bool processed_first_block,
                                          uint64_t in_chan_stride, uint64_t out_chan_stride,
                                          uint64_t pol_stride, unsigned ref_pol, float epsilon)
{
  // there is one complex gain per block
  __shared__ hipFloatComplex g;
  
  // Block X = Channel
  // Block Y = Polarisation
  // Gains are nominally stored in TSPF where T == 1, S == 1, so PF
  //                         ipol       * nchan      + ichan
  const unsigned ipolchan = (blockIdx.y * gridDim.x) + blockIdx.x;

  // get the first thread to read the previously computed complex gain
  if (threadIdx.x == 0)
  {
    g = gains[ipolchan];
  }

  // ensure all threads have the complex gain
  __syncthreads();

  // the reference polarisation could be (unfortunately) not the final ones
  const unsigned ast_pol = (blockIdx.y < ref_pol) ? blockIdx.y : blockIdx.y + 1;

  // offsets for input, reference and output, stored in SFPT
  uint64_t idx = (blockIdx.x * in_chan_stride)  + (ast_pol * pol_stride)    + threadIdx.x;
  uint64_t rdx = (blockIdx.x * in_chan_stride)  + (ref_pol * pol_stride)    + threadIdx.x;
  uint64_t odx = (blockIdx.x * out_chan_stride) + (blockIdx.y * pol_stride) + threadIdx.x;

  __shared__ float normalized_factor;
  float previous_factor = norms[ipolchan];

  for (unsigned iloop=0; iloop<nloops; iloop++)
  {
    // read the reference antenna value
    const hipFloatComplex r = in[rdx];

    // read the astronomy antenna value
    const hipFloatComplex a = in[idx];

    // power in in the astronomy and reference signals
    float pa = blockReduceSumF(squareLawDetect(a));
    float pr = blockReduceSumF(squareLawDetect(r));

    if (threadIdx.x == 0)
    {
      // normalise by the number of values used
      float current_factor = ((pa + pr) / blockDim.x);

      if (processed_first_block || iloop > 0)
        normalized_factor = (0.999 * previous_factor) + (0.001 * current_factor);
      else
        normalized_factor = current_factor;
      previous_factor = normalized_factor;
    }

    // ensure normalized factor is shared across the block
    __syncthreads();

    // compute complex conjugate f = [gain * ref]
    hipFloatComplex f = hipCmulf(g, r);

    // subtract from the astronomy signal [af = ast - f]
    hipFloatComplex af = hipCsubf(a, f);

    // compute correlation [corr = af * conj(ref)]
    hipFloatComplex corr = hipCmulf(af, hipConjf(r));

    corr.x /= normalized_factor;
    corr.y /= normalized_factor;

    // sum corr across the block
    corr = blockReduceSumFC(corr);

    // get thread 0 to compute the new gains
    if (threadIdx.x == 0)
    {
      // normalise
      corr.x /= blockDim.x;
      corr.y /= blockDim.x;

      // compute new gain
      g.x = (corr.x * epsilon) + g.x;
      g.y = (corr.y * epsilon) + g.y;
    }

    // ensure gains are common across the block
    __syncthreads();

    // now that the gain is updated, for this current block
    f = hipCmulf (g, r);

    // and subtract from the astronomy signal
    af = hipCsubf(a, f);

    // write the output global memory
    out[odx] = af;

    //  save dirty and clean signals in first loop only
    if (iloop == nloops-1)
    {
      float cleaned_power_sum = blockReduceSumF(squareLawDetect(af));
      if (threadIdx.x == 0)
      {
        cleaned[ipolchan] = cleaned_power_sum / blockDim.x;
        dirty[ipolchan] = pa / blockDim.x;
        if (blockIdx.y == 0)
        {
          //    npol      * nchan     + ichan
          dirty[gridDim.y * gridDim.x + blockIdx.x] = pr / blockDim.x;
        }
      }
    }
  
    // increment to the next filter
    idx += blockDim.x;
    rdx += blockDim.x;
    odx += blockDim.x;
  }

  // update the gains
  if (threadIdx.x == 0)
  {
    gains[ipolchan] = g;
    norms[ipolchan] = previous_factor;
  }
}


spip::AdaptiveFilterCUDA::AdaptiveFilterCUDA (hipStream_t _stream, string dir) : AdaptiveFilter (dir)
{
  stream = _stream;
  processed_first_block = false;
  gains_file_write = NULL;
  dirty_file_write = NULL;
  cleaned_file_write = NULL;
}

spip::AdaptiveFilterCUDA::~AdaptiveFilterCUDA ()
{
  // ensure the file is closed
  if (gains_file_write)
    gains_file_write->close_file();

  if (dirty_file_write)
    dirty_file_write->close_file();

  if (cleaned_file_write)
    cleaned_file_write->close_file();

  if (gains)
    delete gains;
  gains = NULL;

  if (dirty)
    delete dirty;
  dirty = NULL;

  if (cleaned)
    delete cleaned;
  cleaned = NULL;

  if (norms)
    delete norms;
  norms = NULL;
}

// configure the pipeline prior to runtime
void spip::AdaptiveFilterCUDA::configure (spip::Ordering output_order)
{
  std::string output_dir(".");
  if (!gains)
    gains = new spip::ContainerCUDAFileWrite(stream, output_dir);

  if (!dirty)
    dirty = new spip::ContainerCUDAFileWrite(stream, output_dir);

  if (!cleaned)
    cleaned = new spip::ContainerCUDAFileWrite(stream, output_dir);

  if (!norms)
    norms = new spip::ContainerCUDADevice ();

  spip::AdaptiveFilter::configure (output_order);

  int64_t gains_size = nchan * out_npol * ndim * sizeof(float);
  int64_t dirty_size = nchan * out_npol * sizeof(float);
  int64_t cleaned_size = nchan * out_npol * sizeof(float);

  gains_file_write = dynamic_cast<spip::ContainerCUDAFileWrite *>(gains);
  gains_file_write->set_file_length_bytes (gains_size);
  gains_file_write->process_header ();
  gains_file_write->set_filename_suffix ("gains");

  dirty_file_write = dynamic_cast<spip::ContainerCUDAFileWrite *>(dirty);
  dirty_file_write->set_file_length_bytes (dirty_size);
  dirty_file_write->process_header ();
  dirty_file_write->set_filename_suffix ("dirty");

  cleaned_file_write = dynamic_cast<spip::ContainerCUDAFileWrite *>(cleaned);
  cleaned_file_write->set_file_length_bytes (cleaned_size);
  cleaned_file_write->process_header ();
  cleaned_file_write->set_filename_suffix ("cleaned");
}

// convert to antenna minor order
void spip::AdaptiveFilterCUDA::transform_TSPF()
{
  if (verbose)
    std::cerr << "spip::AdaptiveFilterCUDA::transform_TSPF ()" << endl;
  throw Error (InvalidState, "spip::AdaptiveFilterCUDA::transform_TSPF", "not implemented");
}

void spip::AdaptiveFilterCUDA::transform_SFPT()
{
  if (verbose)
    cerr << "spip::AdaptiveFilterCUDA::transform_SFPT ()" << endl;

  // pointers to the buffers for in, rfi and out
  hipFloatComplex * in = (hipFloatComplex *) input->get_buffer();
  hipFloatComplex * out = (hipFloatComplex *) output->get_buffer();
  hipFloatComplex * gai = (hipFloatComplex *) gains->get_buffer();
  float * dirt = (float *) dirty->get_buffer();
  float * clean = (float *) cleaned->get_buffer();
  float * nor = (float *) norms->get_buffer();

  //dim3 blocks (nsignal, nchan, out_npol);
  dim3 blocks (nchan, out_npol, 1);
  unsigned nthread = 1024;
  if (nthread > filter_update_time)
    nthread = filter_update_time;

  unsigned nloops = ndat / nthread;
  if (ndat % nthread != 0)
    nloops++;

  if (verbose)
  {
    cerr << "spip::AdaptiveFilterCUDA::transform_SFPT nsignal=" << nsignal 
         << " nchan=" << nchan << " npol=" << npol << " out_npol=" << out_npol << " ref_pol=" << ref_pol << endl;
    cerr << "spip::AdaptiveFilterCUDA::transform_SFPT ndat=" << ndat 
          << " nthread=" << nthread << " nloops=" << nloops << " kernels" << endl;
  }

  uint64_t pol_stride = ndat;
  uint64_t in_chan_stride = npol * pol_stride;
  uint64_t in_sig_stride  = nchan * in_chan_stride;
  uint64_t out_chan_stride = out_npol * pol_stride;
  uint64_t out_sig_stride  = nchan * out_chan_stride;

  if (verbose)
    cerr << "spip::AdaptiveFilterCUDA::transform_SFPT blocks=" << blocks.x 
         << "," << blocks.y << "," << blocks.z << " nthread=" << nthread << endl;

  //AdaptiveFilterKernel_SFPT<<<blocks, nthread, 0, stream>>>(in, out, gai, dirt, clean, nor, nloops, processed_first_block, in_sig_stride, in_chan_stride, out_sig_stride, out_chan_stride, pol_stride, ref_pol);
  AdaptiveFilterKernel_FPT<<<blocks, nthread, 0, stream>>>(in, out, gai, dirt, clean, nor, nloops, processed_first_block, in_chan_stride, out_chan_stride, pol_stride, ref_pol, epsilon);
  check_error_stream ("AdaptiveFilterKernel_FPT", stream);

  if (verbose)
    cerr << "spip::AdaptiveFilterCUDA::transform_SFPT kernels complete" << endl;

  processed_first_block = true;
}

// write gains
void spip::AdaptiveFilterCUDA::write_gains ()
{
  uint64_t gains_to_write = (ndat > 0);
  if (verbose)
    cerr << "spip::AdaptiveFilterCUDA::write_gains(" << gains_to_write << ")" << endl;
  gains_file_write->write (gains_to_write);
}

// write dirty
void spip::AdaptiveFilterCUDA::write_dirty ()
{
  uint64_t dirty_to_write = (ndat > 0);
  if (verbose)
    cerr << "spip::AdaptiveFilterCUDA::write_dirty(" << dirty_to_write << ")" << endl;
  dirty_file_write->write (dirty_to_write);
}

// write cleaned
void spip::AdaptiveFilterCUDA::write_cleaned ()
{
  uint64_t cleaned_to_write = (ndat > 0);
  if (verbose)
    cerr << "spip::AdaptiveFilterCUDA::write_cleaned(" << cleaned_to_write << ")" << endl;
  cleaned_file_write->write (cleaned_to_write);
}

