#include "hip/hip_runtime.h"
/***************************************************************************
 *
 *   Copyright (C) 2016 Andrew Jameson
 *   Licensed under the Academic Free License version 2.1
 *
 ***************************************************************************/

#include "spip/ForwardFFTCUDA.h"
#include "spip/CUFFTError.h"

#include <stdexcept>
#include <cmath>

using namespace std;

spip::ForwardFFTCUDA::ForwardFFTCUDA (hipStream_t _stream)
{
  stream = _stream;
  plan = 0;
}

spip::ForwardFFTCUDA::~ForwardFFTCUDA ()
{
  if (plan)
    hipfftDestroy (plan);
  plan = 0;
}

void spip::ForwardFFTCUDA::configure_plan ()
{
  if (plan)
    hipfftDestroy (plan);
  plan = 0;
  
  hipfftResult result = hipfftCreate (&plan);
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "spip::ForwardFFTCUDA::configure_plan",
                      "hipfftCreate(plan)");

  // disable auto-allocation
  hipfftSetAutoAllocation(plan, 0);

  // configure the dimensions for the plan
  configure_plan_dimensions();

  size_t work_area_size;
  result = hipfftMakePlanMany (plan, rank, n,
                              inembed, istride, idist,
                              onembed, ostride, odist,
                              HIPFFT_C2C, howmany, &work_area_size);
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "ForwardFFTCUDA::configure_plan", "hipfftMakePlanMany (plan)");

  result = hipfftSetStream(plan, stream);
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "ForwardFFTCUDA::configure_plan", "hipfftSetStream(plan)");

  // get an estimate on the work buffer size
  work_area_size = 0;
  result = hipfftEstimateMany(rank, n,
                             inembed, istride, idist,
                             onembed, ostride, odist,
                             HIPFFT_C2C, howmany, &work_area_size);
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "ForwardFFTCUDA::configure_plan", "hipfftEstimateMany");

  if (verbose)
    cerr << "ForwardFFTCUDA::configure_plan work_area_size=" << work_area_size << endl;
  auto_allocate = work_area_size > 0;

  if (verbose)
    cerr << "ForwardFFTCUDA::configure_plan auto_allocate=" << auto_allocate << endl;

  //result = hipfftSetAutoAllocation(plan, auto_allocate);
  //if (result != HIPFFT_SUCCESS)
  //  throw CUFFTError (result, "ForwardFFTCUDA::configure_plan", "hipfftSetAutoAllocation");

  if (work_area_size > 0)
  {
    hipError_t error;
    if (work_area)
    {
      error = hipFree (work_area);
      if (error != hipSuccess)
        throw runtime_error("ForwardFFTCUDA::configure_plan hipFree(work_area) failed");
    }

    error = hipMalloc (&work_area, work_area_size);
    if (error != hipSuccess)
      throw runtime_error("ForwardFFTCUDA::configure_plan hipMalloc (work_area) failed");

    result = hipfftSetWorkArea(plan, work_area);
    if (result != HIPFFT_SUCCESS)
      throw CUFFTError (result, "ForwardFFTCUDA::configure_plan", "hipfftSetWorkArea");
  }
  else
    work_area = 0;

}

void spip::ForwardFFTCUDA::prepare ()
{
  spip::ForwardFFT::prepare ();
}

//! perform Forward FFT using CUFFT
void spip::ForwardFFTCUDA::transform_SFPT_to_TFPS ()
{
  hipfftComplex * in  = (hipfftComplex *) input->get_buffer();
  hipfftComplex * out = (hipfftComplex *) output->get_buffer();
  hipfftResult result; 

  uint64_t out_offset;

  for (unsigned isig=0; isig<nsignal; isig++)
  {
    for (unsigned ichan=0; ichan<nchan; ichan++)
    {
      // output channel is offset
      unsigned ochan = ichan * nfft;
      for (unsigned ipol=0; ipol<npol; ipol++)
      {
        out_offset = isig + (ipol * nsignal) + (ochan * npol * nsignal);
        result = hipfftExecC2C(plan, in, out + out_offset, HIPFFT_FORWARD);
        if (result != HIPFFT_SUCCESS)
          throw runtime_error ("ForwardFFTCUDA::tranform hipfftExecC2C failed");

        // increment to next input
        in += ndat;
      }
    }
  }
}

// convert to frequency minor order
void spip::ForwardFFTCUDA::transform_SFPT_to_TSPF ()
{
  if (verbose)
    cerr << "spip::ForwardFFTCUDA::transform_SFPT_to_TSPF()" << endl;
  hipfftComplex * in  = (hipfftComplex *) input->get_buffer();
  hipfftComplex * out = (hipfftComplex *) output->get_buffer();
  hipfftResult result;

  const uint64_t nchan_out = nchan * nfft;
  const uint64_t out_pol_stride = nchan_out;
  const uint64_t out_sig_stride = npol * out_pol_stride;

  // iterate over input ordering of SFPT -> TSPF
  for (unsigned isig=0; isig<nsignal; isig++)
  {
    const uint64_t out_sig_offset = isig * out_sig_stride;
    for (unsigned ichan=0; ichan<nchan; ichan++)
    {
      const uint64_t out_chan_offset = ichan * nfft;
      for (unsigned ipol=0; ipol<npol; ipol++)
      {
        const uint64_t out_pol_offset = ipol * out_pol_stride;

        // process ndat samples, in batches of nfft
        const uint64_t out_offset = out_sig_offset + out_chan_offset + out_pol_offset;

        result = hipfftExecC2C(plan, in, out + out_offset, HIPFFT_FORWARD);
        if (result != HIPFFT_SUCCESS)
          throw runtime_error ("ForwardFFTCUDA::tranform hipfftExecC2C failed");

        in += ndat;
      }
    }
  }
}

void spip::ForwardFFTCUDA::transform_SFPT_to_SFPT ()
{
  hipfftComplex * in  = (hipfftComplex *) input->get_buffer();
  hipfftComplex * out = (hipfftComplex *) output->get_buffer();
  hipfftResult result;

  const uint64_t nchan_out = nchan * nfft;
  const uint64_t out_pol_stride = nbatch;
  const uint64_t out_chan_stride = npol * out_pol_stride;
  const uint64_t out_sig_stride = nchan_out * out_chan_stride;

  // iterate over input ordering of SFPT -> SFPT
  for (unsigned isig=0; isig<nsignal; isig++)
  {
    const uint64_t out_sig_offset = isig * out_sig_stride;
    for (unsigned ichan=0; ichan<nchan; ichan++)
    {
      // output channel will be ichan * nfft
      const uint64_t out_chan_offset = ichan * nfft * out_chan_stride;
      for (unsigned ipol=0; ipol<npol; ipol++)
      {
        const uint64_t out_pol_offset = ipol * out_pol_stride;

        // process ndat samples, in batches of nfft
        const uint64_t out_offset = out_sig_offset + out_chan_offset + out_pol_offset;

        result = hipfftExecC2C(plan, in, out + out_offset, HIPFFT_FORWARD);
        if (result != HIPFFT_SUCCESS)
          throw runtime_error ("ForwardFFTCUDA::tranform hipfftExecC2C failed");

        in += ndat;
      }
    }
  }
}

//! normalize the array by the scale factor
__global__ void forwardfftcuda_normalize_kernel (float * data, uint64_t nval, float scale)
{
  unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < nval)
    data[idx] = data[idx] * scale;
}

void spip::ForwardFFTCUDA::normalize_output ()
{
  float * out = (float *) output->get_buffer();

  int nthread = 1024;
  uint64_t nval = ndat * nsignal * nchan * npol * ndim;
  int nblock = nval / nthread;
  if (nval % nthread != 0)
    nblock++;

  forwardfftcuda_normalize_kernel<<<nblock, nthread, 0, stream>>> (out, nval, scale_fac);

  // TODO add check on running of kernel 
}

