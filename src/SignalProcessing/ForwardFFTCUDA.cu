#include "hip/hip_runtime.h"
/***************************************************************************
 *
 *   Copyright (C) 2016 Andrew Jameson
 *   Licensed under the Academic Free License version 2.1
 *
 ***************************************************************************/

#include "spip/ForwardFFTCUDA.h"
#include "spip/CUFFTError.h"

#include <stdexcept>
#include <cmath>

using namespace std;

spip::ForwardFFTCUDA::ForwardFFTCUDA (hipStream_t _stream)
{
  stream = _stream;
  plan = 0;
}

spip::ForwardFFTCUDA::~ForwardFFTCUDA ()
{
  if (plan)
    hipfftDestroy (plan);
  plan = 0;
}

void spip::ForwardFFTCUDA::configure_plan ()
{
  if (plan)
    hipfftDestroy (plan);
  plan = 0;
  
  hipfftResult result = hipfftCreate (&plan);
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "spip::ForwardFFTCUDA::configure_plan",
                      "hipfftCreate(plan)");

  // disable auto-allocation
  hipfftSetAutoAllocation(plan, 0);

  // configure the dimensions for the plan
  configure_plan_dimensions();

  size_t work_area_size;
  result = hipfftMakePlanMany (plan, rank, n,
                              inembed, istride, idist,
                              onembed, ostride, odist,
                              HIPFFT_C2C, howmany, &work_area_size);
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "ForwardFFTCUDA::configure_plan", "hipfftMakePlanMany (plan)");

  result = hipfftSetStream(plan, stream);
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "ForwardFFTCUDA::configure_plan", "hipfftSetStream(plan)");

  // get an estimate on the work buffer size
  work_area_size = 0;
  result = hipfftEstimateMany(rank, n,
                             inembed, istride, idist,
                             onembed, ostride, odist,
                             HIPFFT_C2C, howmany, &work_area_size);
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "ForwardFFTCUDA::configure_plan", "hipfftEstimateMany");

  if (verbose)
    cerr << "ForwardFFTCUDA::configure_plan work_area_size=" << work_area_size << endl;
  auto_allocate = work_area_size > 0;

  if (verbose)
    cerr << "ForwardFFTCUDA::configure_plan auto_allocate=" << auto_allocate << endl;

  if (work_area_size > 0)
  {
    hipError_t error;
    if (work_area)
    {
      error = hipFree (work_area);
      if (error != hipSuccess)
        throw runtime_error("ForwardFFTCUDA::configure_plan hipFree(work_area) failed");
    }

    error = hipMalloc (&work_area, work_area_size);
    if (error != hipSuccess)
      throw runtime_error("ForwardFFTCUDA::configure_plan hipMalloc (work_area) failed");

    result = hipfftSetWorkArea(plan, work_area);
    if (result != HIPFFT_SUCCESS)
      throw CUFFTError (result, "ForwardFFTCUDA::configure_plan", "hipfftSetWorkArea");
  }
  else
    work_area = 0;
}

__global__ void fftshift_even (float2 *data, int N)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < N)
  {
    const int a = int(1) - (2 * (i & 1));
    data[i].x *= a;
    data[i].y *= a;
  }
}

void spip::ForwardFFTCUDA::fft_shift ()
{
  hipfftComplex * in  = (hipfftComplex *) input->get_buffer();
  hipfftComplex * out = (hipfftComplex *) output->get_buffer();

  uint64_t nval = ndat * nsignal * npol * nchan;
  unsigned nthreads = 1024;
  uint64_t nblocks = nval / nthreads;
  if (nval % nthreads)
    nblocks++;

  if (nfft % 2 == 0)
    fftshift_even<<<nblocks, nthreads, 0, stream>>>(in, nval);
  else
  {
    throw runtime_error ("ForwardFFTCUDA::transform odd fft shift not implemented yet");
  }
}


//! perform Forward FFT using CUFFT
void spip::ForwardFFTCUDA::transform_SFPT_to_TFPS ()
{
  hipfftComplex * in  = (hipfftComplex *) input->get_buffer();
  hipfftComplex * out = (hipfftComplex *) output->get_buffer();
  hipfftResult result; 

  uint64_t out_offset;

  for (unsigned isig=0; isig<nsignal; isig++)
  {
    for (unsigned ichan=0; ichan<nchan; ichan++)
    {
      // output channel is offset
      unsigned ochan = ichan * nfft;
      for (unsigned ipol=0; ipol<npol; ipol++)
      {
        out_offset = isig + (ipol * nsignal) + (ochan * npol * nsignal);
        result = hipfftExecC2C(plan, in, out + out_offset, HIPFFT_FORWARD);
        if (result != HIPFFT_SUCCESS)
          throw runtime_error ("ForwardFFTCUDA::tranform hipfftExecC2C failed");

        // increment to next input
        in += ndat;
      }
    }
  }
}

/*
// kernel to perform fft shift of each FFT block of data
__global__ void forwardfftcuda_fft_shift_tspf_kernel (float2 * output, uint64_t nfft)
{
  // each block of data handles 1 NFFT of data and flips that 
  float2 * data = output + (blockIdx.x * nfft);
  const uint64_t half_nfft = nfft / 2;

  // each thread switches a value with is fft_shifted one
  for (uint64_t i=threadIdx.x; i<half_nfft; i+=blockDim.x)
  {
    float2 upper = data[i + half_nfft];
    data[i + half_nfft] = data[i];
    data[i] = upper;
  }
}
*/

// convert to frequency minor order
void spip::ForwardFFTCUDA::transform_SFPT_to_TSPF ()
{
  if (verbose)
    cerr << "spip::ForwardFFTCUDA::transform_SFPT_to_TSPF()" << endl;
  hipfftComplex * in  = (hipfftComplex *) input->get_buffer();
  hipfftComplex * out = (hipfftComplex *) output->get_buffer();
  hipfftResult result;

  const uint64_t nchan_out = nchan * nfft;
  const uint64_t out_pol_stride = nchan_out;
  const uint64_t out_sig_stride = npol * out_pol_stride;

  // iterate over input ordering of SFPT -> TSPF
  for (unsigned isig=0; isig<nsignal; isig++)
  {
    const uint64_t out_sig_offset = isig * out_sig_stride;
    for (unsigned ichan=0; ichan<nchan; ichan++)
    {
      const uint64_t out_chan_offset = ichan * nfft;
      for (unsigned ipol=0; ipol<npol; ipol++)
      {
        const uint64_t out_pol_offset = ipol * out_pol_stride;

        // process ndat samples, in batches of nfft
        const uint64_t out_offset = out_sig_offset + out_chan_offset + out_pol_offset;

        result = hipfftExecC2C(plan, in, out + out_offset, HIPFFT_FORWARD);
        if (result != HIPFFT_SUCCESS)
          throw runtime_error ("ForwardFFTCUDA::tranform hipfftExecC2C failed");

        in += ndat;
      }
    }
  }

/*
  if (apply_fft_shift)
  {
    // assume that the number of channels dominates
    unsigned nthreads = 1024;
    unsigned half_nfft = nfft / 2;
    if (half_nfft < nthreads)
      nthreads = half_nfft;

    uint64_t ndat_out = ndat / nfft;
    uint64_t nblocks = ndat_out * nsignal * npol * nchan; // nchan is nchan_in

    //cerr << "spip::ForwardFFTCUDA::transform_SFPT_to_TSPF ndat=" << ndat << " ndat_out=" << ndat_out << " nchan=" << nchan << " nfft=" << nfft << endl;
    //cerr << "spip::ForwardFFTCUDA::transform_SFPT_to_TSPF nblocks=" << nblocks << " nthreads=" << nthreads << endl;
    if (verbose)
      cerr << "spip::ForwardFFTCUDA::transform_SFPT_to_TSPF fft_shift nfft=" << nfft << endl;
    forwardfftcuda_fft_shift_tspf_kernel<<<nblocks, nthreads, 0, stream>>> (out, nfft);
  }
*/
}

/*
// kernel to perform fft shift of each FFT block of data
__global__ void forwardfftcuda_fft_shift_sfpt_kernel (float2 * output, uint64_t nfft)
{
  // each block of data handles 1 NFFT of data and flips that 
  float2 * data = output + (blockIdx.x * nfft);
  const uint64_t half_nfft = nfft / 2;
  
  // each thread switches a value with is fft_shifted one
  for (uint64_t i=threadIdx.x; i<half_nfft; i+=blockDim.x)
  { 
    float2 upper = data[i + half_nfft];
    data[i + half_nfft] = data[i];
    data[i] = upper;
  } 
}
*/

void spip::ForwardFFTCUDA::transform_SFPT_to_SFPT ()
{
  hipfftComplex * in  = (hipfftComplex *) input->get_buffer();
  hipfftComplex * out = (hipfftComplex *) output->get_buffer();
  hipfftResult result;

  const uint64_t nchan_out = nchan * nfft;
  const uint64_t out_pol_stride = nbatch;
  const uint64_t out_chan_stride = npol * out_pol_stride;
  const uint64_t out_sig_stride = nchan_out * out_chan_stride;

  // iterate over input ordering of SFPT -> SFPT
  for (unsigned isig=0; isig<nsignal; isig++)
  {
    const uint64_t out_sig_offset = isig * out_sig_stride;
    for (unsigned ichan=0; ichan<nchan; ichan++)
    {
      // output channel will be ichan * nfft
      const uint64_t out_chan_offset = ichan * nfft * out_chan_stride;
      for (unsigned ipol=0; ipol<npol; ipol++)
      {
        const uint64_t out_pol_offset = ipol * out_pol_stride;

        // process ndat samples, in batches of nfft
        const uint64_t out_offset = out_sig_offset + out_chan_offset + out_pol_offset;

        result = hipfftExecC2C(plan, in, out + out_offset, HIPFFT_FORWARD);
        if (result != HIPFFT_SUCCESS)
          throw runtime_error ("ForwardFFTCUDA::tranform hipfftExecC2C failed");

        in += ndat;
      }
    }
  }

/*
  if (apply_fft_shift)
  {
    // assume that the number of channels dominates
    unsigned nthreads = 1024;
    unsigned half_nfft = nfft / 2;

    dim3 blocks = dim3(ndat/nthreads, )

    if (half_nfft < nthreads)
      nthreads = half_nfft;

    uint64_t ndat_out = ndat / nfft;
    uint64_t nblocks = ndat_out * nsignal * npol * nchan; // nchan is nchan_in

    //cerr << "spip::ForwardFFTCUDA::transform_SFPT_to_TSPF ndat=" << ndat << " ndat_out=" << ndat_out << " nchan=" << nchan << " nfft=" << nfft <<
    //endl;
    //cerr << "spip::ForwardFFTCUDA::transform_SFPT_to_TSPF nblocks=" << nblocks << " nthreads=" << nthreads << endl;
    if (verbose)
      cerr << "spip::ForwardFFTCUDA::transform_SFPT_to_SFPT fft_shift nfft=" << nfft << endl;
    forwardfftcuda_fft_shift_sfpt_kernel<<<nblocks, nthreads, 0, stream>>> (out, nfft);
  }
*/
}

//! normalize the array by the scale factor
__global__ void forwardfftcuda_normalize_kernel (float * data, uint64_t nval, float scale)
{
  unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < nval)
    data[idx] = data[idx] * scale;
}

void spip::ForwardFFTCUDA::normalize_output ()
{
  if (verbose)
    cerr << "spip::ForwardFFTCUDA::normalize_output()" << endl;
  float * out = (float *) output->get_buffer();

  int nthread = 1024;
  uint64_t nval = ndat * nsignal * nchan * npol * ndim;
  int nblock = nval / nthread;
  if (nval % nthread != 0)
    nblock++;

  if (verbose)
    cerr << "spip::ForwardFFTCUDA::normalize_output nval="
         << nval << " scale_fac=" << scale_fac << endl;

  forwardfftcuda_normalize_kernel<<<nblock, nthread, 0, stream>>> (out, nval, scale_fac);

  // TODO add check on running of kernel 
}
